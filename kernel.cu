#include <algorithm>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <string>
#include <utility>
#include <vector>

/*
Algorytm jest ten sam, więc bardziej szczegółowe wyjaśnienia są w pliku seq.cpp
(komentarze) Różnią się częsci z symulacja
*/

#define C sqrt(2)
#define SIZE 9
#define NUM_OF_MOVEMENTS_IN_SIMULATION 10
#define MAX_DEPTH 100
#define MOVEMENTS 40
#define MAX_NUMBER_OF_THREADS 1024

enum State { EMPTY, BLACK, WHITE };

std::vector<std::pair<int, int>> NEIGHBOURS[SIZE][SIZE];
State previousPositionForBlack[SIZE][SIZE];
State previousPositionForWhite[SIZE][SIZE];
std::chrono::duration<double> total_time_selection;
std::chrono::duration<double> total_time_expansion;
std::chrono::duration<double> total_time_simulation;
std::chrono::duration<double> total_time_backpropagation;
std::chrono::duration<double> total_time_CPU_GPU_copying;
long selection_moves = 0;
long expansion_moves = 0;
long simulation_moves = 0;
long backpropagation_moves = 0;
int total_taken_black = 0;
int total_taken_white = 0;

typedef struct Node {
  State board[SIZE][SIZE];
  std::vector<Node *> children = std::vector<Node *>();
  Node *parent = NULL;
  int taken_black_stones = 0;
  int taken_white_stones = 0;
  unsigned int number_of_simulations = 0;
  double black_score = 0.0;
  ~Node() {
    for (Node *child : children) {
      delete child;
    }
  }

  std::vector<Node *> getMaxProfitForWhite() {
    std::vector<Node *> profitChildren;
    int profit = taken_black_stones - taken_white_stones;
    int tmp = 0;
    for (Node *child : children) {
      tmp = child->taken_black_stones - child->taken_white_stones;
      if (tmp > profit) {
        profit = tmp;
        profitChildren.clear();
        profitChildren.push_back(child);
      } else if (tmp == profit) {
        profitChildren.push_back(child);
      }
    }
    return profitChildren;
  }

  std::vector<Node *> getMaxProfitForBlack() {
    std::vector<Node *> profitChildren;
    int profit = taken_white_stones - taken_black_stones;
    int tmp = 0;
    for (Node *child : children) {
      tmp = child->taken_white_stones - child->taken_black_stones;
      if (tmp > profit) {
        profit = tmp;
        profitChildren.clear();
        profitChildren.push_back(child);
      } else if (tmp == profit) {
        profitChildren.push_back(child);
      }
    }
    return profitChildren;
  }
} Node;

double calculateUct(Node *n, State state) {
  if (n->number_of_simulations == 0) {
    return std::numeric_limits<double>::infinity();
  }
  double w_i = 0.0;
  if (state == BLACK) {
    w_i = n->black_score;
  } else {
    w_i = (double)n->number_of_simulations - n->black_score;
  }
  return (w_i / n->number_of_simulations) +
         C * sqrt(log(n->parent->number_of_simulations) /
                  n->number_of_simulations);
}

void copyBoard(State source[SIZE][SIZE], State destination[SIZE][SIZE]) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      destination[i][j] = source[i][j];
    }
  }
}

void printBoard(State board[SIZE][SIZE]) {
  std::cout << "    ";
  for (int i = 0; i < SIZE; ++i) {
    std::cout << i << ' ';
  }
  std::cout << '\n';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << i << ' ' << ' ' << " ";
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == EMPTY) {
        std::cout << ". ";
      } else if (board[i][j] == BLACK) {
        std::cout << "X ";
      } else {
        std::cout << "O ";
      }
    }
    std::cout << '\n';
  }
  std::cout << '\n';
}

bool isKo(State prev[SIZE][SIZE], State actual[SIZE][SIZE]) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (prev[i][j] != actual[i][j])
        return false;
    }
  }
  return true;
}

void createNeighbours() {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      std::vector<std::pair<int, int>> neighbours;
      if (i - 1 >= 0) {
        neighbours.push_back(std::make_pair(i - 1, j));
      }
      if (i + 1 < SIZE) {
        neighbours.push_back(std::make_pair(i + 1, j));
      }
      if (j - 1 >= 0) {
        neighbours.push_back(std::make_pair(i, j - 1));
      }
      if (j + 1 < SIZE) {
        neighbours.push_back(std::make_pair(i, j + 1));
      }
      NEIGHBOURS[i][j] = neighbours;
    }
  }
}

std::pair<std::vector<std::pair<int, int>>, std::vector<std::pair<int, int>>>
findReached(State board[SIZE][SIZE], int i, int j) {
  State color = board[i][j];
  std::vector<std::pair<int, int>> chain, reached;
  std::vector<std::pair<int, int>> frontier = {std::make_pair(i, j)};
  while (!frontier.empty()) {
    auto current_fc = frontier.back();
    frontier.pop_back();
    if (std::find(chain.begin(), chain.end(), current_fc) == chain.end()) {
      chain.push_back(current_fc);
    }
    for (auto fn : NEIGHBOURS[current_fc.first][current_fc.second]) {
      if (board[fn.first][fn.second] == color &&
          std::find(chain.begin(), chain.end(), fn) == chain.end()) {
        frontier.push_back(fn);
      } else if (board[fn.first][fn.second] != color &&
                 std::find(reached.begin(), reached.end(), fn) ==
                     reached.end()) {
        reached.push_back(fn);
      }
    }
  }
  return std::make_pair(chain, reached);
}

State changeState(State state) {
  if (state == BLACK) {
    return WHITE;
  } else {
    return BLACK;
  }
}

std::pair<bool, std::vector<std::pair<int, int>>>
couldPlaceStone(State board[SIZE][SIZE], int row, int col, State state) {
  std::vector<std::pair<int, int>> taken_stones =
      std::vector<std::pair<int, int>>();
  if (board[row][col] != EMPTY)
    return std::make_pair(false, taken_stones);
  board[row][col] = state;
  State alternativeState = changeState(state);

  std::vector<std::pair<int, int>> stones_reached_by_mine =
      findReached(board, row, col).second;
  // tu sprawdzamy czy cos zabieramy u przeciwnika
  for (int i = 0; i < stones_reached_by_mine.size(); ++i) {
    if (board[stones_reached_by_mine[i].first]
             [stones_reached_by_mine[i].second] !=
        EMPTY) { // dla kazego kamienia z sąsiadujących
      auto tmp_chain_reached =
          findReached(board, stones_reached_by_mine[i].first,
                      stones_reached_by_mine[i].second);
      auto potential_captured = tmp_chain_reached.first;
      auto reached_by_opponent = tmp_chain_reached.second;
      int j = 0;
      for (j = 0; j < reached_by_opponent.size();
           ++j) { // dla kazdego wezla co jest obok kamienia przeciwnika
        if (board[reached_by_opponent[j].first]
                 [reached_by_opponent[j].second] != state) {
          break;
        }
      }
      if (j == reached_by_opponent.size()) {
        for (int ind = 0; ind < potential_captured.size(); ++ind) {
          taken_stones.push_back(std::make_pair(
              potential_captured[ind].first, potential_captured[ind].second));
        }
      }
    }
  }
  // end
  // jak nic nie zabieramy to czy nie ma samobojstwa
  if (taken_stones.size() == 0) {
    int j = 0;
    for (j = 0; j < stones_reached_by_mine.size(); ++j) {
      if (board[stones_reached_by_mine[j].first]
               [stones_reached_by_mine[j].second] != alternativeState) {
        break;
      }
    }
    if (j == stones_reached_by_mine
                 .size()) { // only opponent stones surround me, suicide
      board[row][col] = EMPTY;
      return std::make_pair(false, taken_stones);
    }
  }
  // KO check
  bool is_ko;
  for (int i = 0; i < taken_stones.size(); ++i) {
    board[taken_stones[i].first][taken_stones[i].second] = EMPTY;
  }
  if (state == WHITE) {
    is_ko = isKo(previousPositionForBlack, board);
  } else {
    is_ko = isKo(previousPositionForWhite, board);
  }
  if (is_ko) {
    return std::make_pair(false, taken_stones);
  }

  for (int i = 0; i < taken_stones.size(); ++i) {
    board[taken_stones[i].first][taken_stones[i].second] = alternativeState;
  }
  board[row][col] = EMPTY;
  return std::make_pair(true, taken_stones);
}

void createChildren(Node *n, int i, int j, State state) {
  State tmp_board[SIZE][SIZE];
  copyBoard(n->board, tmp_board);
  auto taken_stones = couldPlaceStone(tmp_board, i, j, state).second;
  Node *childNode = new Node();
  copyBoard(n->board, childNode->board);
  childNode->board[i][j] = state;
  childNode->taken_black_stones = n->taken_black_stones;
  childNode->taken_white_stones = n->taken_white_stones;
  for (int i = 0; i < taken_stones.size(); ++i) {
    childNode->board[taken_stones[i].first][taken_stones[i].second] = EMPTY;
  }
  if (state == BLACK) {
    childNode->taken_white_stones =
        childNode->taken_white_stones + taken_stones.size();
  } else {
    childNode->taken_black_stones =
        childNode->taken_black_stones + taken_stones.size();
  }
  n->children.push_back(childNode);
  childNode->parent = n;
}

std::pair<int, int> computeTerritories(State board[SIZE][SIZE]) {
  bool managed[SIZE][SIZE];
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      managed[i][j] = false;
    }
  }
  std::vector<std::pair<int, int>> chain;
  std::vector<std::pair<int, int>> reached;
  State color;
  int white_territory = 0;
  int black_territory = 0;
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == EMPTY && !managed[i][j]) {
        auto res = findReached(board, i, j);
        chain = res.first;
        reached = res.second;
        for (auto p : chain) {
          managed[p.first][p.second] = true;
        }
        // jesli w reached wszystkie kamienie sa jednego koloru to to jest
        // terytorium tego koloru
        color = board[reached[0].first][reached[0].second];
        int k = 0;
        for (k = 0; k < reached.size(); ++k) {
          if (board[reached[k].first][reached[k].second] != color) {
            break;
          }
        }
        if (k == reached.size()) {
          if (color == BLACK) {
            black_territory += chain.size();
          } else {
            white_territory += chain.size();
          }
        }
      }
    }
  }
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == BLACK) {
        ++black_territory;
      } else if (board[i][j] == WHITE) {
        ++white_territory;
      }
    }
  }
  return std::make_pair(black_territory, white_territory);
}

void expand(Node *n, State state) {
  State tmp_board[SIZE][SIZE];
  copyBoard(n->board, tmp_board);
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (couldPlaceStone(tmp_board, i, j, state).first) {
        createChildren(
            n, i, j,
            state); // i j - tu ustawimy kamien i takie dziecko dodamy do n
      }
    }
  }
}

void flattenCube(Node *n, State *flattenedCube) {
  int index = 0;
  for (int k = 0; k < n->children.size();
       ++k) { // tworzymy kostkę żeby potem przesłać plansze wszystkich dzieci
              // na GPU
    for (int i = 0; i < SIZE; ++i) {
      for (int j = 0; j < SIZE; ++j) {
        flattenedCube[index] = n->children[k]->board[i][j];
        ++index;
      }
    }
  }
}

// analogia findReached, tylko zamiast wektora par używam 2-wymiarowych tablic
// na wejsciu chain i reached są wypełnione -1 (dla łatwiejszego pilnowania
// rozmiaru)
__device__ void d_findReached(State board[SIZE][SIZE], int i, int j,
                              int chain[SIZE * SIZE][2],
                              int reached[SIZE * SIZE][2]) {
  State color = board[i][j];
  int frontier[SIZE * SIZE][2] = {{i, j}};
  int frontierSize = 1;
  int current_fc[2];
  bool in_chain[SIZE][SIZE];
  bool in_reached[SIZE][SIZE];
  int chain_size = 0;
  int reached_size = 0;
  for (int i = 0; i < SIZE * SIZE; ++i) {
    in_chain[i][0] = false;
    in_chain[i][1] = false;
    in_reached[i][0] = false;
    in_reached[i][1] = false;
  }
  while (frontierSize != 0) {
    current_fc[0] = frontier[frontierSize - 1][0];
    current_fc[1] = frontier[frontierSize - 1][1];
    --frontierSize;
    if (!in_chain[current_fc[0]][current_fc[1]]) {
      chain[chain_size][0] = current_fc[0];
      chain[chain_size][1] = current_fc[1];
      in_chain[current_fc[0]][current_fc[1]] = true;
      ++chain_size;
    }
    int i = current_fc[0];
    int j = current_fc[1];

    if (i - 1 >= 0) {
      if (board[i - 1][j] == color && !in_chain[i - 1][j]) {
        frontier[frontierSize][0] = i - 1;
        frontier[frontierSize][1] = j;
        ++frontierSize;
      } else if (board[i - 1][j] != color && in_reached[i - 1][j]) {
        reached[reached_size][0] = i - 1;
        reached[reached_size][1] = j;
        in_reached[i - 1][j] = true;
        ++reached_size;
      }
    }
    if (i + 1 < SIZE) {
      if (board[i + 1][j] == color && !in_chain[i + 1][j]) {
        frontier[frontierSize][0] = i + 1;
        frontier[frontierSize][1] = j;
        ++frontierSize;
      } else if (board[i + 1][j] != color && in_reached[i + 1][j]) {
        reached[reached_size][0] = i + 1;
        reached[reached_size][1] = j;
        in_reached[i + 1][j] = true;
        ++reached_size;
      }
    }
    if (j - 1 >= 0) {
      if (board[i][j - 1] == color && !in_chain[i][j - 1]) {
        frontier[frontierSize][0] = i;
        frontier[frontierSize][1] = j - 1;
        ++frontierSize;
      } else if (board[i][j - 1] != color && in_reached[i][j - 1]) {
        reached[reached_size][0] = i;
        reached[reached_size][1] = j - 1;
        in_reached[i][j - 1] = true;
        ++reached_size;
      }
    }
    if (j + 1 < SIZE) {
      if (board[i][j + 1] == color && !in_chain[i][j + 1]) {
        frontier[frontierSize][0] = i;
        frontier[frontierSize][1] = j + 1;
        ++frontierSize;
      } else if (board[i][j + 1] != color && in_reached[i][j + 1]) {
        reached[reached_size][0] = i;
        reached[reached_size][1] = j + 1;
        in_reached[i][j + 1] = true;
        ++reached_size;
      }
    }
  }
}

// taken_stones jest wypełnione -1
__device__ bool d_couldPlaceStone(State board[SIZE][SIZE], int row, int col,
                                  State state,
                                  int taken_stones[SIZE * SIZE][2]) {
  int taken_stones_size = 0;
  if (board[row][col] != EMPTY) {
    return false;
  }
  board[row][col] = state;
  State alternativeState = BLACK;
  if (state == BLACK) {
    alternativeState = WHITE;
  }

  int chain[SIZE * SIZE][2];
  int reached[SIZE * SIZE][2];
  for (int i = 0; i < SIZE * SIZE; ++i) {
    chain[i][0] = -1;
    chain[i][1] = -1;
    reached[i][0] = -1;
    reached[i][1] = -1;
  }

  d_findReached(board, row, col, chain, reached);
  int r_ind = 0;
  while (reached[r_ind][0] != -1) {
    int potential_captured[SIZE * SIZE][2];
    int reached_by_opponent[SIZE * SIZE][2];
    for (int i = 0; i < SIZE * SIZE; ++i) {
      potential_captured[i][0] = -1;
      potential_captured[i][1] = -1;
      reached_by_opponent[i][0] = -1;
      reached_by_opponent[i][1] = -1;
    }

    if (board[reached[r_ind][0]][reached[r_ind][1]] != EMPTY) {
      d_findReached(board, reached[r_ind][0], reached[r_ind][1],
                    potential_captured, reached_by_opponent);
      int j = 0;
      while (reached_by_opponent[j][0] != -1) {
        if (board[reached_by_opponent[j][0]][reached_by_opponent[j][1]] !=
            state) {
          break;
        }
        ++j;
      }
      if (reached_by_opponent[j][0] == -1) {
        int ind = 0;
        while (potential_captured[ind][0] != -1) {
          taken_stones[taken_stones_size][0] = potential_captured[ind][0];
          taken_stones[taken_stones_size][1] = potential_captured[ind][1];
          ++taken_stones_size;
          ++ind;
        }
      }
    }
    ++r_ind;
  }

  if (taken_stones[0][0] == -1) {
    int j = 0;
    while (reached[j][0] != -1) {
      if (board[reached[j][0]][reached[j][1]] != alternativeState) {
        break;
      }
      ++j;
    }
    if (reached[j][0] == -1) {
      board[row][col] = EMPTY;
      return false;
    }
  }
  board[row][col] = EMPTY;
  return true;
}

__device__ void d_computeTerritories(State board[SIZE][SIZE], int results[2]) {
  bool managed[SIZE][SIZE];
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      managed[i][j] = false;
    }
  }
  int chain[SIZE * SIZE][2];
  int reached[SIZE * SIZE][2];
  State color;
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      for (int ind = 0; ind < SIZE * SIZE; ++ind) {
        chain[ind][0] = -1;
        reached[ind][0] = -1;
        chain[ind][1] = -1;
        reached[ind][1] = -1;
      }
      if (board[i][j] == EMPTY && !managed[i][j]) {
        d_findReached(board, i, j, chain, reached);
        int chain_size = 0;
        while (chain[chain_size][0] >= 0 && chain[chain_size][0] < SIZE &&
               chain[chain_size][1] >= 0 && chain[chain_size][1] < SIZE) {
          managed[chain[chain_size][0]][chain[chain_size][1]] = true;
          ++chain_size;
        }
        color = board[reached[0][0]][reached[0][1]];
        int tmp = 0;
        while (reached[tmp][0] != -1) {
          if (board[reached[tmp][0]][reached[tmp][1]] != color) {
            break;
          }
          ++tmp;
        }
        if (reached[tmp][0] == -1) {
          if (color == BLACK) {
            results[0] += chain_size;
          } else {
            results[1] += chain_size;
          }
        }
      }
    }
  }

  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == BLACK) {
        results[0] = results[0] + 1;
      } else if (board[i][j] == WHITE) {
        results[1] = results[1] + 1;
      }
    }
  }
}

__global__ void randomPlaysKernel(
    State *d_flattenedCubes,
    double *d_black_scores, // kazdy blok wpisze liczbe punktów czarnych po
                            // wszystkich symulacjach dla odpowiadającego diecka
    int *d_taken_black_stones, int *d_taken_white_stones,
    State *state_in_simulation) {
  __shared__ double
      localCounts[MAX_NUMBER_OF_THREADS]; // kazdy wątek zazanacza w swojej
                                          // komórce wynik symulacji dla
                                          // czarnych (1, 0.5 lub 0)
  int taken_stones[SIZE * SIZE][2];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState cs;
  hiprand_init(clock64() + tid, 0, 0, &cs);
  State board_for_random_play[SIZE][SIZE];
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      board_for_random_play[i][j] =
          d_flattenedCubes[SIZE * SIZE * blockIdx.x + SIZE * i + j];
    }
  }
  State state = BLACK;
  if (*state_in_simulation == BLACK) {
    state = WHITE;
  }

  int num_of_tries = 0;
  int index = 0;
  int lost_black_stones = d_taken_black_stones[blockIdx.x];
  int lost_white_stones = d_taken_white_stones[blockIdx.x];
  int random_row, random_col;
  bool could_place_stone;

  while (index < NUM_OF_MOVEMENTS_IN_SIMULATION) {
    for (int i = 0; i < SIZE * SIZE; ++i) {
      taken_stones[i][0] = -1;
      taken_stones[i][1] = -1;
    }
    do {
      ++num_of_tries;
      random_row = hiprand(&cs) % SIZE;
      random_col = hiprand(&cs) % SIZE;

      could_place_stone = d_couldPlaceStone(board_for_random_play, random_row,
                                            random_col, state, taken_stones);
    } while (!could_place_stone && num_of_tries < 20);
    if (num_of_tries == 20) {
      break;
    }
    int num_of_taken_stones = 0;
    while (taken_stones[num_of_taken_stones][0] != -1) {
      ++num_of_taken_stones;
    }

    board_for_random_play[random_row][random_col] = state;
    if (state == BLACK) {
      lost_white_stones += num_of_taken_stones;
    } else {
      lost_black_stones += num_of_taken_stones;
    }
    if (state == BLACK) {
      state = WHITE;
    } else {
      state = BLACK;
    }
    int ind = 0;
    while (taken_stones[ind][0] != -1) {
      board_for_random_play[taken_stones[ind][0]][taken_stones[ind][1]] = EMPTY;
      ++ind;
    }
    num_of_tries = 0;
    ++index;
  }

  int results[2] = {0, 0};
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board_for_random_play[i][j] == BLACK) {
        results[0] = results[0] + 1;
      } else if (board_for_random_play[i][j] == WHITE) {
        results[1] = results[1] + 1;
      }
    }
  }
  if (state == WHITE) {
    results[1] = results[1] + 1;
  }
  //__syncthreads();
  if ((results[0] + lost_white_stones) > (results[1] + lost_black_stones)) {
    localCounts[threadIdx.x] = 1.0;
  } else if ((results[0] + lost_white_stones) ==
             (results[1] + lost_black_stones)) {
    localCounts[threadIdx.x] = 0.5;
  }
  __syncthreads();
  if (threadIdx.x ==
      0) { // jeden główny wątek, który zapisze wuynik od całego bloku
    for (int i = 0; i < MAX_NUMBER_OF_THREADS; ++i) {
      d_black_scores[blockIdx.x] += localCounts[i];
    }
  }
}

void simulate(Node *n, State state) {
  hipError_t cudaStatus;
  int totalSize = n->children.size() * SIZE * SIZE;
  State *h_flattenedCubes = new State[totalSize];
  flattenCube(n, h_flattenedCubes);
  State *d_flattenedCubes;
  cudaStatus = hipMalloc(&d_flattenedCubes, totalSize * sizeof(State));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_flattenedCubes) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }

  std::chrono::high_resolution_clock::time_point start, end;
  start = std::chrono::high_resolution_clock::now();
  cudaStatus = hipMemcpy(d_flattenedCubes, h_flattenedCubes,
                          totalSize * sizeof(State), hipMemcpyHostToDevice);
  end = std::chrono::high_resolution_clock::now();
  total_time_CPU_GPU_copying +=
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_flattenedCubes) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  double *h_black_scores = new double[n->children.size()];
  double *d_black_scores; // kazde dziecko ma NUM_OF_THREADS
                          // symulacji, tu kazde zapisze liczba
                          // wygranych dla czarnych
  cudaStatus = hipMalloc(&d_black_scores, n->children.size() * sizeof(double));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_black_scores) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  cudaStatus =
      hipMemset(d_black_scores, 0, n->children.size() * sizeof(double));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemset (d_black_scores) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  // state_in_simulation = state;
  State *d_state;
  cudaStatus = hipMalloc(&d_state, sizeof(State));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_state) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  start = std::chrono::high_resolution_clock::now();
  cudaStatus =
      hipMemcpy(d_state, &state, sizeof(State), hipMemcpyHostToDevice);
  end = std::chrono::high_resolution_clock::now();
  total_time_CPU_GPU_copying +=
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_state) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  int *h_taken_white_stones = new int[n->children.size()];
  int *d_taken_white_stones;
  cudaStatus = hipMalloc((void **)&d_taken_white_stones,
                          n->children.size() * sizeof(int));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_taken_white_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  int *h_taken_black_stones = new int[n->children.size()];
  int *d_taken_black_stones;
  cudaStatus = hipMalloc((void **)&d_taken_black_stones,
                          n->children.size() * sizeof(int));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_taken_black_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  for (int i = 0; i < n->children.size(); ++i) {
    h_taken_black_stones[i] = n->children[i]->taken_black_stones;
    h_taken_white_stones[i] =
        n->children[i]->taken_white_stones; // juz zdobyte kamienie
  }
  start = std::chrono::high_resolution_clock::now();
  cudaStatus =
      hipMemcpy(d_taken_white_stones, h_taken_white_stones,
                 n->children.size() * sizeof(int), hipMemcpyHostToDevice);

  end = std::chrono::high_resolution_clock::now();
  total_time_CPU_GPU_copying +=
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_taken_white_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  start = std::chrono::high_resolution_clock::now();
  cudaStatus =
      hipMemcpy(d_taken_black_stones, h_taken_black_stones,
                 n->children.size() * sizeof(int), hipMemcpyHostToDevice);
  end = std::chrono::high_resolution_clock::now();
  total_time_CPU_GPU_copying +=
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_taken_black_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }

  randomPlaysKernel<<<n->children.size(), MAX_NUMBER_OF_THREADS>>>(
      d_flattenedCubes, d_black_scores, d_taken_black_stones,
      d_taken_white_stones, d_state);
  hipDeviceSynchronize();
  start = std::chrono::high_resolution_clock::now();
  hipMemcpy(h_black_scores, d_black_scores,
             n->children.size() * sizeof(double), hipMemcpyDeviceToHost);
  end = std::chrono::high_resolution_clock::now();
  total_time_CPU_GPU_copying +=
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  for (int i = 0; i < n->children.size(); ++i) {
    n->number_of_simulations += MAX_NUMBER_OF_THREADS;
    n->children[i]->number_of_simulations += MAX_NUMBER_OF_THREADS;
    n->black_score = n->black_score + h_black_scores[i];
    n->children[i]->black_score =
        n->children[i]->black_score + h_black_scores[i];
  }
  delete[] h_flattenedCubes;
  delete[] h_black_scores;
  delete[] h_taken_black_stones;
  delete[] h_taken_white_stones;
  hipFree(d_flattenedCubes);
  hipFree(d_black_scores);
  hipFree(d_taken_black_stones);
  hipFree(d_taken_white_stones);
  hipFree(d_state);
}

Node *findMaxUctChild(Node *parent, State state) {
  double maxUCT = -std::numeric_limits<double>::infinity();
  Node *maxUCTChild = parent->children[0];
  std::vector<Node *> topChildren = std::vector<Node *>();
  if (state == BLACK) {
    topChildren = parent->getMaxProfitForBlack();
  } else {
    topChildren = parent->getMaxProfitForWhite();
  }
  for (Node *child : topChildren) {
    double uctValue = calculateUct(child, state);
    if (uctValue > maxUCT) {
      maxUCT = uctValue;
      maxUCTChild = child;
    }
  }
  return maxUCTChild;
}

void backpropagate(Node *n) {
  Node *tmp = n;
  while (tmp->parent) {
    tmp->parent->black_score += tmp->black_score;
    tmp->parent->number_of_simulations += tmp->number_of_simulations;
    tmp = tmp->parent;
  }
}

Node *makeHumanMove(Node *parent, State state, int i, int j) {
  for (Node *child : parent->children) {
    if (child->board[i][j] == state) {
      return child;
    }
  }
  std::cout << "Helloo\n";
  return parent->children[0];
}

void showResults(Node *root_node, State actual_state) {
  std::cout << "Now we will see the results\n";
  std::cout << "Previous position for black:\n";
  printBoard(previousPositionForBlack);
  std::cout << "Previous position for white:\n";
  printBoard(previousPositionForWhite);
  if (actual_state == BLACK) {
    copyBoard(previousPositionForBlack, root_node->board);
  } else {
    copyBoard(previousPositionForWhite, root_node->board);
  }
  std::cout << "Main board\n";
  printBoard(root_node->board);
  auto main_results = computeTerritories(root_node->board);
  std::cout << "\nBlack territory: " << main_results.first << '\n';
  std::cout << "White territory: " << main_results.second << '\n';
  int lost_black_stones = total_taken_black;
  int lost_white_stones = total_taken_white;
  std::cout << "Lost black stones: " << total_taken_black << '\n';
  std::cout << "Lost white stones: " << total_taken_white << '\n';
  std::cout << "-------------------------------------\n";
  if ((main_results.first + lost_white_stones) >
      (main_results.second + lost_black_stones)) {
    std::cout << "BLACK won\n";
  } else if ((main_results.first + lost_white_stones) <
             (main_results.second + lost_black_stones)) {
    std::cout << "WHITE won\n";
  } else {
    std::cout << "DRAW\n";
  }
  std::cout << "-------------------------------------\n";
}

void play(Node *root_node, State actual_state, bool isHumanVsComp,
          State humanState) {
  Node *actual_node;
  State whoose_move = actual_state;
  int max_depth_ind = 0;
  int mov_ind = 0;
  int row_by_user, col_by_user;
  std::cout << "\nStart board: \n";
  printBoard(root_node->board);
  std::chrono::high_resolution_clock::time_point start, end;
  while (mov_ind < MOVEMENTS) {
    max_depth_ind = 0;
    while (max_depth_ind < MAX_DEPTH) {
      actual_node = root_node;
      int local_depth = 0;
      while (actual_node->children.size() != 0) {
        if (local_depth % 2 == 0) {
          whoose_move = actual_state;
        } else {
          whoose_move = changeState(actual_state);
        }
        start = std::chrono::high_resolution_clock::now();
        actual_node = findMaxUctChild(actual_node, whoose_move); // select
        end = std::chrono::high_resolution_clock::now();
        ++selection_moves;
        total_time_selection +=
            std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        ++local_depth;
      }
      if (local_depth % 2 == 0) {
        whoose_move = actual_state;
      } else {
        whoose_move = changeState(actual_state);
      }
      start = std::chrono::high_resolution_clock::now();
      expand(actual_node, whoose_move);
      end = std::chrono::high_resolution_clock::now();
      ++expansion_moves;
      total_time_expansion +=
          std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
      start = std::chrono::high_resolution_clock::now();
      simulate(actual_node, whoose_move);
      end = std::chrono::high_resolution_clock::now();
      ++simulation_moves;
      total_time_simulation +=
          std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
      start = std::chrono::high_resolution_clock::now();
      backpropagate(actual_node);
      end = std::chrono::high_resolution_clock::now();
      ++backpropagation_moves;
      total_time_backpropagation +=
          std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
      ++max_depth_ind;
    }

    if (isHumanVsComp && actual_state == humanState) {
      std::cout << "Your move:\n";
      std::cin >> row_by_user >> col_by_user;
      if (row_by_user == -1)
        return;
      root_node =
          makeHumanMove(root_node, actual_state, row_by_user, col_by_user);
    } else {
      root_node = findMaxUctChild(root_node,
                                  actual_state);
    }

    std::cout << "\nNr: " << mov_ind << '\n';
    printBoard(root_node->board);
    total_taken_black = root_node->taken_black_stones;
    total_taken_white = root_node->taken_white_stones;
    std::cout << "Lost black stones: " << root_node->taken_black_stones << '\n';
    std::cout << "Lost white stones: " << root_node->taken_white_stones << '\n';
    if (actual_state == BLACK) { // przekazujemy prawo ruchu innemy graczowi
      actual_state = WHITE;
      copyBoard(root_node->board, previousPositionForWhite);
    } else {
      actual_state = BLACK;
      copyBoard(root_node->board, previousPositionForBlack);
    }
    ++mov_ind;
  }
}

void emptyBoard(State actual_board[SIZE][SIZE]) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      actual_board[i][j] = EMPTY;
    }
  }
}

void preProcessing(Node *root_node, State &actual_state,
                   State actual_board[SIZE][SIZE], bool &is_black,
                   bool &isHumanVsComp, State &humanState, bool is_load_board,
                   std::string &filename) {
  std::srand(std::time(0));
  emptyBoard(actual_board);
  createNeighbours();

  int num_of_o = 0;
  int num_of_x = 0;
  if (is_load_board) {
    std::ifstream file(filename);
    if (file.is_open()) {
      std::string line;
      int i = 0;
      while (getline(file, line)) {
        int j = 0;
        for (char c : line) {
          if (c == 'o' || c == 'O') {
            actual_board[i][j] = WHITE;
            ++num_of_o;
          } else if (c == 'x' || c == 'X') {
            actual_board[i][j] = BLACK;
            ++num_of_x;
          }
          ++j;
        }
        ++i;
      }

      file.close();
    } else {
      std::cerr << "Unable to open file: " << filename << std::endl;
      exit(1);
    }
  }
  if (num_of_o != num_of_x) {
    actual_state = WHITE;
  }

  copyBoard(actual_board, root_node->board);
  std::cout << "Input board:\n";
  printBoard(root_node->board);
  if (actual_state == BLACK) {
    copyBoard(actual_board, previousPositionForBlack);
    emptyBoard(previousPositionForWhite);
  } else {
    copyBoard(actual_board, previousPositionForWhite);
    emptyBoard(previousPositionForBlack);
  }

  int tmp;
  std::cout
      << "Select mode:\n 1 - copmuter vs computer\n 2 - human vs computer\n";
  std::cin >> tmp;
  if (tmp == 2) {
    isHumanVsComp = true;
  }
  if (isHumanVsComp) {
    std::cout << "Select color:\n 1 - black\n 2 - white\n";
    std::cin >> tmp;
    if (tmp == 2) {
      humanState = WHITE;
    }
  }
}

void showTime() {
  double average_time_selection =
      total_time_selection.count() / (double)selection_moves;
  double average_time_expansion =
      total_time_expansion.count() / (double)expansion_moves;
  double average_time_simulation =
      total_time_simulation.count() / (double)simulation_moves;
  double average_time_backpropagation =
      total_time_backpropagation.count() / (double)backpropagation_moves;
  std::cout << "Selection average time: " << average_time_selection << " \n";
  std::cout << "Expansion average time: " << average_time_expansion << " \n";
  std::cout << "Simulation average time: " << average_time_simulation << " \n";
  std::cout << "Backpropagation average time: " << average_time_backpropagation
            << " \n";
  std::cout << "CPU-GPU data copying: " << total_time_CPU_GPU_copying.count()
            << '\n';
}

int main(int argc, char **argv) {
  State actual_state = BLACK;
  bool is_black = true;
  State actual_board[SIZE][SIZE];
  Node *root_node = new Node;
  bool isHumanVsComp = false;
  bool is_load_board = false;
  std::string filename = "";
  if (argc >= 2) {
    is_load_board = true;
    filename = argv[1];
  }
  State humanState = BLACK;
  preProcessing(root_node, actual_state, actual_board, is_black, isHumanVsComp,
                humanState, is_load_board, filename);
  play(root_node, actual_state, isHumanVsComp, humanState);
  showResults(root_node, actual_state);
  showTime();
  delete root_node;
  return 0;
}
