#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <string>
#include <utility>
#include <vector>

#define C sqrt(2)
#define SIZE 9
#define NUM_OF_MOVEMENTS_IN_SIMULATION 10
#define MAX_DEPTH 5 // tyle razy wykonamy te 3 etapy
#define MOVEMENTS 6
#define MAX_NUMBER_OF_THREADS 256

enum State { EMPTY, BLACK, WHITE };

//__device__ __managed__ State state_in_simulation;

std::vector<std::pair<int, int>> NEIGHBOURS[SIZE][SIZE];
State previousPositionForBlack[SIZE][SIZE];
State previousPositionForWhite[SIZE][SIZE];

typedef struct Node {
  State board[SIZE][SIZE];
  std::vector<Node *> children = std::vector<Node *>();
  Node *parent = NULL;
  int taken_black_stones = 0;
  int taken_white_stones = 0;
  unsigned int number_of_simulations = 0;
  double black_score = 0.0;
  // double uct;
  ~Node() {
    for (Node *child : children) {
      delete child;
    }
  }

  std::vector<Node *> getMaxProfitForWhite() {
    std::vector<Node *> profitChildren;
    int profit = taken_black_stones - taken_white_stones;
    int tmp = 0;
    for (Node *child : children) {
      tmp = child->taken_black_stones - child->taken_white_stones;
      if (tmp > profit) {
        profit = tmp;
        profitChildren.clear();
        profitChildren.push_back(child);
      } else if (tmp == profit) {
        profitChildren.push_back(child);
      }
    }
    return profitChildren;
  }

  std::vector<Node *> getMaxProfitForBlack() {
    std::vector<Node *> profitChildren;
    int profit = taken_white_stones - taken_black_stones;
    int tmp = 0;
    for (Node *child : children) {
      tmp = child->taken_white_stones - child->taken_black_stones;
      if (tmp > profit) {
        profit = tmp;
        profitChildren.clear();
        profitChildren.push_back(child);
      } else if (tmp == profit) {
        profitChildren.push_back(child);
      }
    }
    return profitChildren;
  }
} Node;

double calculateUct(Node *n, State state) {
  if (n->number_of_simulations == 0) {
    return std::numeric_limits<double>::
        infinity(); // Return infinity if the child has not been explored yet.
  }
  double w_i = 0.0;
  if (state == BLACK) {
    w_i = n->black_score;
  } else {
    w_i = (double)n->number_of_simulations - n->black_score;
  }
  return (w_i / n->number_of_simulations) +
         C * sqrt(log(n->parent->number_of_simulations) /
                  n->number_of_simulations);
}

void copyBoard(State source[SIZE][SIZE], State destination[SIZE][SIZE]) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      destination[i][j] = source[i][j];
    }
  }
}

void printBoard(Node *n) {
  std::cout << '\t';
  std::cout << '\t';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << i << '\t';
  }
  std::cout << '\n';
  std::cout << '\t';
  std::cout << '\t';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << "_\t";
  }
  std::cout << '\n';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << i << '\t' << '|' << '\t';
    for (int j = 0; j < SIZE; ++j) {
      if (n->board[i][j] == EMPTY) {
        std::cout << ".\t";
      } else if (n->board[i][j] == BLACK) {
        std::cout << "X\t";
      } else {
        std::cout << "O\t";
      }
    }
    std::cout << '\n';
  }
  std::cout << '\n';
}

void printPrevPosBoard(State board[SIZE][SIZE]) {
  std::cout << '\t';
  std::cout << '\t';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << i << '\t';
  }
  std::cout << '\n';
  std::cout << '\t';
  std::cout << '\t';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << "_\t";
  }
  std::cout << '\n';
  for (int i = 0; i < SIZE; ++i) {
    std::cout << i << '\t' << '|' << '\t';
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == EMPTY) {
        std::cout << ".\t";
      } else if (board[i][j] == BLACK) {
        std::cout << "X\t";
      } else {
        std::cout << "O\t";
      }
    }
    std::cout << '\n';
  }
  std::cout << '\n';
}


bool isKo(State prev[SIZE][SIZE], State actual[SIZE][SIZE]) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (prev[i][j] != actual[i][j])
        return false;
    }
  }
  return true;
}

void generateRandomCell(int &randomRow, int &randomCol) {
  randomRow = std::rand() % SIZE;
  randomCol = std::rand() % SIZE;
}

void createNeighbours() {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      std::vector<std::pair<int, int>> neighbours;
      if (i - 1 >= 0) {
        neighbours.push_back(std::make_pair(i - 1, j));
      }
      if (i + 1 < SIZE) {
        neighbours.push_back(std::make_pair(i + 1, j));
      }
      if (j - 1 >= 0) {
        neighbours.push_back(std::make_pair(i, j - 1));
      }
      if (j + 1 < SIZE) {
        neighbours.push_back(std::make_pair(i, j + 1));
      }
      NEIGHBOURS[i][j] = neighbours;
    }
  }
}

std::pair<std::vector<std::pair<int, int>>, std::vector<std::pair<int, int>>>
findReached(State board[SIZE][SIZE], int i, int j) {
  State color = board[i][j];
  std::vector<std::pair<int, int>> chain, reached;
  std::vector<std::pair<int, int>> frontier = {std::make_pair(i, j)};
  while (!frontier.empty()) {
    auto current_fc = frontier.back();
    frontier.pop_back();
    if (std::find(chain.begin(), chain.end(), current_fc) == chain.end()) {
      chain.push_back(current_fc);
    }
    for (auto fn : NEIGHBOURS[current_fc.first][current_fc.second]) {
      if (board[fn.first][fn.second] == color &&
          std::find(chain.begin(), chain.end(), fn) == chain.end()) {
        frontier.push_back(fn);
      } else if (board[fn.first][fn.second] != color &&
                 std::find(reached.begin(), reached.end(), fn) ==
                     reached.end()) {
        reached.push_back(fn);
      }
    }
  }
  return std::make_pair(chain, reached);
}

State changeState(State state) {
  if (state == BLACK) {
    return WHITE;
  } else {
    return BLACK;
  }
}

std::pair<bool, std::vector<std::pair<int, int>>>
couldPlaceStone(State board[SIZE][SIZE], int row, int col, State state) {
  std::vector<std::pair<int, int>> taken_stones =
      std::vector<std::pair<int, int>>();
  if (board[row][col] != EMPTY)
    return std::make_pair(false, taken_stones);
  board[row][col] = state;
  State alternativeState = changeState(state);

  std::vector<std::pair<int, int>> stones_reached_by_mine =
      findReached(board, row, col).second;
  // tu sprawdzamy czy cos zabieramy u przeciwnika
  for (int i = 0; i < stones_reached_by_mine.size(); ++i) {
    if (board[stones_reached_by_mine[i].first]
             [stones_reached_by_mine[i].second] !=
        EMPTY) { // dla kazego kamienia z sąsiadujących
      auto tmp_chain_reached =
          findReached(board, stones_reached_by_mine[i].first,
                      stones_reached_by_mine[i].second);
      auto potential_captured = tmp_chain_reached.first;
      auto reached_by_opponent = tmp_chain_reached.second;
      int j = 0;
      for (j = 0; j < reached_by_opponent.size();
           ++j) { // dla kazdego wezla co jest obok kamienia przeciwnika
        if (board[reached_by_opponent[j].first]
                 [reached_by_opponent[j].second] != state) {
          break;
        }
      }
      if (j == reached_by_opponent.size()) {
        for (int ind = 0; ind < potential_captured.size(); ++ind) {
          taken_stones.push_back(std::make_pair(
              potential_captured[ind].first, potential_captured[ind].second));
        }
      }
    }
  }
  // end
  // jak nic nie zabieramy to czy nie ma samobojstwa
  if (taken_stones.size() == 0) {
    int j = 0;
    for (j = 0; j < stones_reached_by_mine.size(); ++j) {
      if (board[stones_reached_by_mine[j].first]
               [stones_reached_by_mine[j].second] != alternativeState) {
        break;
      }
    }
    if (j == stones_reached_by_mine
                 .size()) { // only opponent stones surround me, suicide
      board[row][col] = EMPTY;
      return std::make_pair(false, taken_stones);
    }
  }
  if (state == BLACK) {
    isKo(previousPositionForBlack, board);
  } else {
    isKo(previousPositionForWhite, board);
  }
  board[row][col] = EMPTY;
  return std::make_pair(true, taken_stones);
}

void createChildren(Node *n, int i, int j, State state) {
  auto taken_stones = couldPlaceStone(n->board, i, j, state).second;
  Node *childNode = new Node();
  copyBoard(n->board, childNode->board);
  childNode->board[i][j] = state;
  childNode->taken_black_stones = n->taken_black_stones;
  childNode->taken_white_stones = n->taken_white_stones;
  for (int i = 0; i < taken_stones.size(); ++i) {
    childNode->board[taken_stones[i].first][taken_stones[i].second] = EMPTY;
  }
  if (state == BLACK) {
    childNode->taken_white_stones =
        childNode->taken_white_stones + taken_stones.size();
  } else {
    childNode->taken_black_stones =
        childNode->taken_black_stones + taken_stones.size();
  }
  n->children.push_back(childNode);
  childNode->parent = n;
}

std::pair<int, int> computeTerritories(State board[SIZE][SIZE]) {
  bool managed[SIZE][SIZE];
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      managed[i][j] = false;
    }
  }
  std::vector<std::pair<int, int>> chain;
  std::vector<std::pair<int, int>> reached;
  State color;
  int white_territory = 0;
  int black_territory = 0;
  
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == EMPTY && !managed[i][j]) {
        auto res = findReached(board, i, j);
        std::cout<<"i = "<<i<<" j = "<<j<<'\n';
        chain = res.first;
        reached = res.second;
        for (auto p : chain) {
          managed[p.first][p.second] = true;
        }
        // jesli w reached wszystkie kamienie sa jednego koloru to to jest
        // terytorium tego koloru
        color = board[reached[0].first][reached[0].second];
        int k = 0;
        for (k = 0; k < reached.size(); ++k) {
          if (board[reached[k].first][reached[k].second] != color) {
            break;
          }
        }
        if (k == reached.size()) {
          if (color == BLACK) {
            black_territory += chain.size();
          } else {
            white_territory += chain.size();
          }
        }
      }
    }
  }
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == BLACK) {
        ++black_territory;
      } else if (board[i][j] == WHITE) {
        ++white_territory;
      }
    }
  }
  return std::make_pair(black_territory, white_territory);
}

void expand(Node *n, State state) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (couldPlaceStone(n->board, i, j, state).first) {
        createChildren(
            n, i, j,
            state); // i j - tu ustawimy kamien i takie dziecko dodamy do n
      }
    }
  }
}

void flattenCube(Node *n, State *flattenedCube) {
  int index = 0;
  for (int k = 0; k < n->children.size(); ++k) {
    for (int i = 0; i < SIZE; ++i) {
      for (int j = 0; j < SIZE; ++j) {
        flattenedCube[index] = n->children[k]->board[i][j];
        ++index;
      }
    }
  }
}

__device__ void d_findReached(State board[SIZE][SIZE], int i, int j,
                              int chain[SIZE * SIZE][2],
                              int reached[SIZE * SIZE][2]) {
  State color = board[i][j];
  int frontier[SIZE * SIZE][2] = {{i, j}};
  int frontierSize = 1;
  int current_fc[2];
  bool in_chain[SIZE][SIZE];
  bool in_reached[SIZE][SIZE];
  int chain_size = 0;
  int reached_size = 0;
  for (int i = 0; i < SIZE * SIZE; ++i) {
    in_chain[i][0] = false;
    in_chain[i][1] = false;
    in_reached[i][0] = false;
    in_reached[i][1] = false;
  }
  while (frontierSize != 0) {
    current_fc[0] = frontier[frontierSize - 1][0];
    current_fc[1] = frontier[frontierSize - 1][1];
    --frontierSize;
    if (!in_chain[current_fc[0]][current_fc[1]]) {
      chain[chain_size][0] = current_fc[0];
      chain[chain_size][1] = current_fc[1];
      in_chain[current_fc[0]][current_fc[1]] = true;
      ++chain_size;
    }
    int i = current_fc[0];
    int j = current_fc[1];

    if (i - 1 >= 0) {
      if (board[i - 1][j] == color && !in_chain[i - 1][j]) {
        frontier[frontierSize][0] = i - 1;
        frontier[frontierSize][1] = j;
        ++frontierSize;
      } else if (board[i - 1][j] != color && in_reached[i - 1][j]) {
        reached[reached_size][0] = i - 1;
        reached[reached_size][1] = j;
        in_reached[i - 1][j] = true;
        ++reached_size;
      }
    }
    if (i + 1 < SIZE) {
      if (board[i + 1][j] == color && !in_chain[i + 1][j]) {
        frontier[frontierSize][0] = i + 1;
        frontier[frontierSize][1] = j;
        ++frontierSize;
      } else if (board[i + 1][j] != color && in_reached[i + 1][j]) {
        reached[reached_size][0] = i + 1;
        reached[reached_size][1] = j;
        in_reached[i + 1][j] = true;
        ++reached_size;
      }
    }
    if (j - 1 >= 0) {
      if (board[i][j - 1] == color && !in_chain[i][j - 1]) {
        frontier[frontierSize][0] = i;
        frontier[frontierSize][1] = j - 1;
        ++frontierSize;
      } else if (board[i][j - 1] != color && in_reached[i][j - 1]) {
        reached[reached_size][0] = i;
        reached[reached_size][1] = j - 1;
        in_reached[i][j - 1] = true;
        ++reached_size;
      }
    }
    if (j + 1 < SIZE) {
      if (board[i][j + 1] == color && !in_chain[i][j + 1]) {
        frontier[frontierSize][0] = i;
        frontier[frontierSize][1] = j + 1;
        ++frontierSize;
      } else if (board[i][j + 1] != color && in_reached[i][j + 1]) {
        reached[reached_size][0] = i;
        reached[reached_size][1] = j + 1;
        in_reached[i][j + 1] = true;
        ++reached_size;
      }
    }
  }
}
// taken_stones, reached. chain sa wypelnione -1
__device__ bool d_couldPlaceStone(State board[SIZE][SIZE], int row, int col,
                                  State state,
                                  int taken_stones[SIZE * SIZE][2]) {
  int taken_stones_size = 0;
  if (board[row][col] != EMPTY) {
    return false;
  }
  board[row][col] = state;
  State alternativeState = BLACK;
  if (state == BLACK) {
    alternativeState = WHITE;
  }

  int chain[SIZE * SIZE][2];
  int reached[SIZE * SIZE][2];
  for (int i = 0; i < SIZE * SIZE; ++i) {
    chain[i][0] = -1;
    chain[i][1] = -1;
    reached[i][0] = -1;
    reached[i][1] = -1;
  }

  d_findReached(board, row, col, chain, reached);
  int r_ind = 0;
  while (reached[r_ind][0] != -1) {
    int potential_captured[SIZE * SIZE][2];
    int reached_by_opponent[SIZE * SIZE][2];
    for (int i = 0; i < SIZE * SIZE; ++i) {
      potential_captured[i][0] = -1;
      potential_captured[i][1] = -1;
      reached_by_opponent[i][0] = -1;
      reached_by_opponent[i][1] = -1;
    }

    if (board[reached[r_ind][0]][reached[r_ind][1]] != EMPTY) {
      d_findReached(board, reached[r_ind][0], reached[r_ind][1],
                    potential_captured, reached_by_opponent);
      int j = 0;
      while (reached_by_opponent[j][0] != -1) {
        if (board[reached_by_opponent[j][0]][reached_by_opponent[j][1]] !=
            state) {
          break;
        }
        ++j;
      }
      if (reached_by_opponent[j][0] == -1) {
        int ind = 0;
        while (potential_captured[ind][0] != -1) {
          taken_stones[taken_stones_size][0] = potential_captured[ind][0];
          taken_stones[taken_stones_size][1] = potential_captured[ind][1];
          ++taken_stones_size;
          ++ind;
        }
      }
    }
    ++r_ind;
  }

  if (taken_stones[0][0] == -1) {
    int j = 0;
    while (reached[j][0] != -1) {
      if (board[reached[j][0]][reached[j][1]] != alternativeState) {
        break;
      }
      ++j;
    }
    if (reached[j][0] == -1) {
      board[row][col] = EMPTY;
      return false;
    }
  }
  // is Ko - TO DO
  board[row][col] = EMPTY;
  return true;
}

__device__ void d_computeTerritories(State board[SIZE][SIZE], int results[2]) {
  bool managed[SIZE][SIZE];
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      managed[i][j] = false;
    }
  }
  int chain[SIZE * SIZE][2];
  int reached[SIZE * SIZE][2];
  State color;
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      for (int ind = 0; ind < SIZE * SIZE; ++ind) {
        chain[ind][0] = -1;
        reached[ind][0] = -1;
        chain[ind][1] = -1;
        reached[ind][1] = -1;
      }
      if (board[i][j] == EMPTY && !managed[i][j]) {
        d_findReached(board, i, j, chain, reached);
        int chain_size = 0;
        while (chain[chain_size][0] >= 0 && chain[chain_size][0] < SIZE &&
               chain[chain_size][1] >= 0 &&
               chain[chain_size][1] < SIZE /*chain[chain_size][0] != -1*/) {
          managed[chain[chain_size][0]][chain[chain_size][1]] =
              true; // cudamemcheck
          ++chain_size;
        }
        color = board[reached[0][0]][reached[0][1]];
        // int k = 0;
        int tmp = 0;
        while (reached[tmp][0] != -1) {
          if (board[reached[tmp][0]][reached[tmp][1]] != color) {
            break;
          }
          ++tmp;
        }
        if (reached[tmp][0] == -1) {
          if (color == BLACK) {
            results[0] += chain_size;
          } else {
            results[1] += chain_size;
          }
        }
      }
    }
  }

  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      if (board[i][j] == BLACK) {
        results[0] = results[0] + 1;
      } else if (board[i][j] == WHITE) {
        results[1] = results[1] + 1;
      }
    }
  }
}

__global__ void
randomPlaysKernel(State *d_flattenedCubes,
                  int *d_black_scores,       // out
                  int *d_taken_black_stones, // just info for point counting
                  int *d_taken_white_stones, State *state_in_simulation) {

  int taken_stones[SIZE * SIZE][2];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // printf("Hello from kernel\n");
  hiprandState cs;
  hiprand_init(clock64(), tid, 0, &cs);
  State board_for_random_play[SIZE][SIZE];
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      board_for_random_play[i][j] =
          d_flattenedCubes[SIZE * SIZE * blockIdx.x + SIZE * i + j];
    }
  }
  //__syncthreads(); // Synchronize threads to ensure all data is copied
  State state = BLACK;
  if (*state_in_simulation == BLACK) {
    state = WHITE;
  }

  int num_of_tries = 0;
  int index = 0;
  int lost_black_stones = d_taken_black_stones[blockIdx.x];
  int lost_white_stones = d_taken_white_stones[blockIdx.x];
  int random_row, random_col;
  bool could_place_stone;

  while (index < NUM_OF_MOVEMENTS_IN_SIMULATION) {

    for (int i = 0; i < SIZE * SIZE; ++i) {
      taken_stones[i][0] = -1;
      taken_stones[i][1] = -1;
    }
    do {
      ++num_of_tries;
      random_row = hiprand(&cs) % SIZE;
      random_col = hiprand(&cs) % SIZE;

      could_place_stone = d_couldPlaceStone(board_for_random_play, random_row,
                                            random_col, state, taken_stones);
    } while (!could_place_stone && num_of_tries < 20);
    if (num_of_tries == 20) {
      break;
    }
    int num_of_taken_stones = 0;
    while (taken_stones[num_of_taken_stones][0] != -1) {
      ++num_of_taken_stones;
    }

    board_for_random_play[random_row][random_col] = state;
    if (state == BLACK) {
      lost_white_stones += num_of_taken_stones;
    } else {
      lost_black_stones += num_of_taken_stones;
    }
    if (state == BLACK) {
      state = WHITE;
    } else {
      state = BLACK;
    }
    int ind = 0;
    while (taken_stones[ind][0] != -1) {
      board_for_random_play[taken_stones[ind][0]][taken_stones[ind][1]] = EMPTY;
      ++ind;
    }
    num_of_tries = 0;
    ++index;
  }

  int results[2] = {0, 0};
  d_computeTerritories(board_for_random_play, results);
  if ((results[0] + lost_white_stones) > (results[1] + lost_black_stones)) {
    atomicAdd(&d_black_scores[blockIdx.x], 1);
  }
  // if(black win) atomicadd(d_black_scores[blockIdx.x], 1)
}

void simulate(Node *n, State state) {
  hipError_t cudaStatus;
  int totalSize = n->children.size() * SIZE * SIZE;
  State *h_flattenedCubes = new State[totalSize];
  flattenCube(n, h_flattenedCubes);
  State *d_flattenedCubes;
  cudaStatus = hipMalloc(&d_flattenedCubes, totalSize * sizeof(State));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_flattenedCubes) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  cudaStatus = hipMemcpy(d_flattenedCubes, h_flattenedCubes,
                          totalSize * sizeof(State), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_flattenedCubes) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  int *h_black_scores =
      new int[n->children.size()]; // kazde dziecko ma 1024 symulacji, tu
                                   // kazde zapisze liczba wygranych dla
                                   // czarnych
  int *d_black_scores;
  cudaStatus = hipMalloc(&d_black_scores, n->children.size() * sizeof(int));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_black_scores) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  cudaStatus = hipMemset(d_black_scores, 0, n->children.size() * sizeof(int));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemset (d_black_scores) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  // state_in_simulation = state;
  State *d_state;
  cudaStatus = hipMalloc(&d_state, sizeof(State));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_state) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  cudaStatus =
      hipMemcpy(d_state, &state, sizeof(State), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_state) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  int *h_taken_white_stones = new int[n->children.size()];
  int *d_taken_white_stones;
  cudaStatus = hipMalloc((void **)&d_taken_white_stones,
                          n->children.size() * sizeof(int));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_taken_white_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  int *h_taken_black_stones = new int[n->children.size()];
  int *d_taken_black_stones;
  cudaStatus = hipMalloc((void **)&d_taken_black_stones,
                          n->children.size() * sizeof(int));
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMalloc (d_taken_black_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  } 
  for (int i = 0; i < n->children.size(); ++i) {
    h_taken_black_stones[i] = n->children[i]->taken_black_stones;
    h_taken_white_stones[i] =
        n->children[i]->taken_white_stones; // juz zdobyte kamienie
  }
  cudaStatus =
      hipMemcpy(d_taken_white_stones, h_taken_white_stones,
                 n->children.size() * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_taken_white_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  } 
  cudaStatus =
      hipMemcpy(d_taken_black_stones, h_taken_black_stones,
                 n->children.size() * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    std::cout << "[ERROR] hipMemcpy (d_taken_black_stones) failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    exit(1);
  }
  
  randomPlaysKernel<<<n->children.size(), MAX_NUMBER_OF_THREADS>>>(
      d_flattenedCubes, d_black_scores, d_taken_black_stones,
      d_taken_white_stones, d_state);
  hipDeviceSynchronize();

  hipMemcpy(h_black_scores, d_black_scores, n->children.size() * sizeof(int),
             hipMemcpyDeviceToHost);
  for (int i = 0; i < n->children.size(); ++i) {
    n->number_of_simulations += MAX_NUMBER_OF_THREADS;
    n->children[i]->number_of_simulations += MAX_NUMBER_OF_THREADS;
    n->black_score = n->black_score + h_black_scores[i];
    n->children[i]->black_score =
        n->children[i]->black_score + h_black_scores[i];
  }
  delete[] h_flattenedCubes;
  delete[] h_black_scores;
  delete[] h_taken_black_stones;
  delete[] h_taken_white_stones;
  hipFree(d_flattenedCubes);
  hipFree(d_black_scores);
  hipFree(d_taken_black_stones);
  hipFree(d_taken_white_stones);
  hipFree(d_state);
}

Node *findMaxUctChild(Node *parent, State state) {
  double maxUCT = -std::numeric_limits<double>::infinity();
  Node *maxUCTChild = nullptr;
  std::vector<Node *> topChildren = std::vector<Node *>();
  if (state == BLACK) {
    topChildren = parent->getMaxProfitForBlack();
  } else {
    topChildren = parent->getMaxProfitForWhite();
  }
  for (Node *child : topChildren) {
    double uctValue = calculateUct(child, state);
    if (uctValue > maxUCT) {
      maxUCT = uctValue;
      maxUCTChild = child;
    }
  }
  return maxUCTChild;
}

void backpropagate(Node *n) {
  Node *tmp = n;
  while (tmp->parent) {
    tmp->parent->black_score += tmp->black_score;
    tmp->parent->number_of_simulations += tmp->number_of_simulations;
    tmp = tmp->parent;
  }
}

Node *makeHumanMove(Node *parent, State state, int i, int j) {
  for (Node *child : parent->children) {
    if (child->board[i][j] == state) {
      return child;
    }
  }
  return parent->children[0];
}

void showResults(Node *root_node, State actual_state) {
  std::cout<<"Hello from show results\n";
  printBoard(root_node);
  std::cout<<"Previous position for black:\n";
  printPrevPosBoard(previousPositionForBlack);
  std::cout<<"Previous position for white:\n";
  printPrevPosBoard(previousPositionForWhite);
  if(actual_state == BLACK){
    copyBoard(previousPositionForBlack, root_node->board);
  } else {
  copyBoard(previousPositionForWhite, root_node->board);
  }
  auto main_results = computeTerritories(root_node->board);
  std::cout << "\nBlack territory: " << main_results.first << '\n';
  std::cout << "White territory: " << main_results.second << '\n';
  int lost_black_stones = root_node->taken_black_stones;
  int lost_white_stones = root_node->taken_white_stones;
  std::cout << "Lost black stones: " << lost_black_stones << '\n';
  std::cout << "Lost white stones: " << lost_white_stones << '\n';
  if ((main_results.first + lost_white_stones) >
      (main_results.second + lost_black_stones)) {
    std::cout << "BLACK won\n";
  } else if ((main_results.first + lost_white_stones) <
             (main_results.second + lost_black_stones)) {
    std::cout << "WHITE won\n";
  } else {
    std::cout << "DRAW\n";
  }
}

void play(Node *root_node, State actual_state, bool isHumanVsComp,
          State humanState) {
  Node *actual_node;
  State whoose_move = actual_state;
  int max_depth_ind = 0;
  int mov_ind = 0;
  int row_by_user, col_by_user;
  std::cout << "\nStart board: \n";
  printBoard(root_node);
  while (mov_ind < MOVEMENTS) {
    max_depth_ind = 0;
    while (max_depth_ind < MAX_DEPTH) {
      actual_node = root_node;
      int local_depth = 0;
      while (actual_node->children.size() != 0) {
        if (local_depth % 2 == 0) {
          whoose_move = actual_state;
        } else {
          whoose_move = changeState(actual_state);
        }
        actual_node = findMaxUctChild(actual_node, whoose_move); // select
        ++local_depth;
      }
      if (local_depth % 2 == 0) {
        whoose_move = actual_state;
      } else {
        whoose_move = changeState(actual_state);
      }
      expand(actual_node, whoose_move);
      simulate(actual_node, whoose_move);
      backpropagate(actual_node);
      ++max_depth_ind;
    }

    if (isHumanVsComp && actual_state == humanState) {
      std::cout << "Your move:\n";
      std::cin >> row_by_user >> col_by_user;
      root_node =
          makeHumanMove(root_node, actual_state, row_by_user, col_by_user);
    } else {
      root_node = findMaxUctChild(root_node,
                                  actual_state); // na pewno bo tu robimy ruch
    }

    std::cout << "\nNr: " << mov_ind << '\n';
    printBoard(root_node);
    std::cout << "Lost black stones: " << root_node->taken_black_stones << '\n';
    std::cout << "Lost white stones: " << root_node->taken_white_stones << '\n';
    if (actual_state == BLACK) { // przekazujemy prawo ruchu innemy graczowi
      actual_state = WHITE;
      copyBoard(root_node->board, previousPositionForWhite);
    } else {
      actual_state = BLACK;
      copyBoard(root_node->board, previousPositionForBlack);
    }
    ++mov_ind;
  }
}

void emptyBoard(State actual_board[SIZE][SIZE]) {
  for (int i = 0; i < SIZE; ++i) {
    for (int j = 0; j < SIZE; ++j) {
      actual_board[i][j] = EMPTY;
    }
  }
}

void loadBoard(State &actual_state, State actual_board[SIZE][SIZE]) {
  std::cout << "Load your board\n";
  std::cin.ignore();
  std::string input;
  int num_of_x = 0;
  int num_of_o = 0;
  for (int i = 0; i < SIZE; ++i) {
    std::getline(std::cin, input);
    int char_nr = 0;
    for (char c : input) {
      if (c == 'X' || c == 'x') {
        ++num_of_x;
        actual_board[i][char_nr] = BLACK;
      } else if (c == 'O' || c == 'o') {
        ++num_of_o;
        actual_board[i][char_nr] = WHITE;
      }
      ++char_nr;
    }
  }
  if (num_of_x != num_of_o) {
    actual_state = WHITE;
  }
}

void preProcessing(Node *root_node, State &actual_state,
                   State actual_board[SIZE][SIZE], bool &is_black,
                   bool &isHumanVsComp, State &humanState) {
  std::srand(std::time(0));
  emptyBoard(actual_board);
  createNeighbours();
  int tmp;
  std::cout << "Do you want to load board? 1 - yes, 2 - no\n";
  std::cin >> tmp;
  if (tmp == 1) {
    loadBoard(actual_state, actual_board);
  }
  copyBoard(actual_board, root_node->board);
  copyBoard(actual_board, previousPositionForBlack);
  copyBoard(actual_board, previousPositionForWhite);

  std::cout
      << "Select mode:\n 1 - copmuter vs computer\n 2 - human vs computer\n";
  std::cin >> tmp;
  if (tmp == 2) {
    isHumanVsComp = true;
  }
  if (isHumanVsComp) {
    std::cout << "Select color:\n 1 - black\n 2 - white\n";
    std::cin >> tmp;
    if (tmp == 2) {
      humanState = WHITE;
    }
  }
}

int main(int argc, char **argv) {
  State actual_state = BLACK;
  bool is_black = true;
  State actual_board[SIZE][SIZE];
  Node *root_node = new Node;
  bool isHumanVsComp = false;
  State humanState = BLACK;
  preProcessing(root_node, actual_state, actual_board, is_black, isHumanVsComp,
                humanState);
  play(root_node, actual_state, isHumanVsComp, humanState);
  std::cout<<"Now we will see results\n";
  showResults(root_node, actual_state);
  //delete[] root_node;
  return 0;
}
